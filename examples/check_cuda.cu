
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

int main()
{
	hipDeviceProp_t prop;
	hipError_t status;
	int device_count;
	status = hipGetDeviceCount(&device_count);
	int device_index = 0;
	if (status != hipSuccess) { 
		fprintf(stderr,"cudaGetDeviceCount() failed: %s\n", hipGetErrorString(status)); 
		return -1;
	}
	if (device_index >= device_count) {
		fprintf(stderr, "Specified device index %d exceeds the maximum (the device count on this system is %d)\n", device_index, device_count);
		return -1;
	}
	status = hipGetDeviceProperties(&prop, device_index);
	if (status != hipSuccess) { 
		fprintf(stderr,"cudaGetDeviceProperties() for device ${device_index} failed: %s\n", hipGetErrorString(status)); 
		return -1;
	}
	printf("%d%d %d.%d", prop.major, prop.minor, prop.major, prop.minor);
	return 0;
}
